#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        #define blockSize 128
        int* obuffer;
        int* ibuffer;


        __global__ void upSweep(int n, int* idata, int layer) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

			int skip = 1 << (layer + 1); // powf(2, layer + 1);

            int i = (index) * skip;
            if (i + skip - 1 >= n) {
                return;
            }
          
           
            idata[int(i + skip - 1)] += idata[int(i + (skip >> 1) - 1)];

        
        }

        __global__ void downSweep(int n, int* idata, int layer) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
          
   
			int skip = 1 << (layer + 1); // powf(2, layer + 1);
            int i = index * skip;
            if (i + skip - 1 >= n) {
                return;
			}   
          


            int t = idata[int(i + (skip >> 1) - 1)];
            
        
            idata[int(i + (skip >> 1) - 1)] = idata[int(i + skip - 1)];
            idata[int(i + skip - 1)] += t;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO

            int size = 1 << ilog2ceil(n); 
       
            int* padded = new int[size];

            for (int i = 0; i < size; i++) {
                if (i < n) {
                    padded[i] = idata[i];
                }
                else {
					padded[i] = 0;
                }
                
            }

            int* obuffer;
            int* ibuffer;
            hipMalloc((void**)&obuffer, size * sizeof(int));
            hipMalloc((void**)&ibuffer, size * sizeof(int));

      
            hipMemcpy(obuffer, padded, size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(ibuffer, padded, size * sizeof(int), hipMemcpyHostToDevice);
            hipDeviceSynchronize();

           
          
        
          
			// up sweep
	
            int numBlocks;


            for (int layer = 0; layer <= ilog2ceil(size) - 1; layer++) {
                int numThreads = size / int(powf(2, layer + 1));
                if (numThreads == 0) continue;

				numBlocks = (numThreads + blockSize - 1) / blockSize;
                upSweep<<<numBlocks, blockSize>>>(size, ibuffer, layer);
                hipDeviceSynchronize();
               
             
            }

	        
			hipMemset(ibuffer + size - 1, 0, sizeof(int));
         
         

           // down sweep 
   
         
            for (int layer = ilog2ceil(size) - 1; layer >= 0; layer--) {
                int numThreads = size / int(powf(2, layer + 1));
                if (numThreads == 0) continue;

                numBlocks = (numThreads + blockSize - 1) / blockSize;
                downSweep<<<numBlocks, blockSize>>>(size, ibuffer, layer);
                hipDeviceSynchronize();
               

			}
            timer().endGpuTimer();



   
            hipMemcpy(odata, ibuffer, n * sizeof(int), hipMemcpyDeviceToHost);
         
         
			delete[] padded;
			padded = nullptr;
			hipFree(ibuffer);
			hipFree(obuffer);
            
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
          //  timer().startGpuTimer();
            // TODO
            int count = 0;
            int* flags = new int[n];
            int* scanned = new int[n];
            int* temp = new int[n];
            for (int i = 0; i < n; i++) {
                if (idata[i] != 0) {
                    flags[i] = 1;
                }
                else {
                    flags[i] = 0;
                }
            }

			scan(n, scanned, flags);


            for (int i = 0; i < n; i++) {
                if (flags[i] == 1) {
                    temp[scanned[i]] = idata[i];
					count++;
                }

			}
            for (int i = 0; i < count; i++) {
                odata[i] = temp[i];
            }


         //   timer().endGpuTimer();
            return count;
        }
    }
}
